#include <stdbool.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main ( void ){
if (argc < 2) {
		printf("Required args: N(dimension), k\n");
		exit(-1);
	}
int N = atoi(argv[2]);
hipError_t cudaStat ; // hipMalloc status
hipblasStatus_t stat ; // CUBLAS functions status
hipblasHandle_t handle ; // CUBLAS context

float *h_a, *h_b, *h_c,*h_cc;
        // Memory Allocation in Host
h_a = (float *)malloc(nBytes);
h_b = (float *)malloc(nBytes);
h_c = (float *)malloc(nBytes);
h_cc = (float *)malloc(nBytes);
// define an mxk matrix a column by column

for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
        h_a[IDX2C(i,j,N)] = 1.0;
        h_b[IDX2C(i,j,N)] = 2.0;
        h_c[IDX2C(i,j,N)] = 0.0;
      }
    }

float *d_a, *d_b, *d_c;
hipMalloc((void **) &d_a, sizeof(float)*N*N);
hipMalloc((void **) &d_b, sizeof(float)*N*N);
hipMalloc((void **) &d_c, sizeof(float)*N*N);

stat = hipblasCreate (& handle ); // initialize CUBLAS context
// copy matrices from the host to the device
stat = hipblasSetMatrix (N,N, sizeof (*h_a),h_a,N,d_a ,N); //a -> d_a
stat = hipblasSetMatrix (N,N, sizeof (*h_b),h_b,N,d_b ,N); //b -> d_b
stat = hipblasSetMatrix (N,N, sizeof (*h_c),h_c,N,d_c ,N); //c -> d_c
float al =1.0f; // al =1
float bet =1.0f; // bet =1
// matrix - matrix multiplication : d_c = al*d_a *d_b + bet *d_c
// d_a -mxk matrix , d_b -kxn matrix , d_c -mxn matrix ;
// al ,bet -scalars
stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,,N,N,N,&al,d_a,N,d_b,N,&bet,d_c,N);
stat = hipblasGetMatrix (N,N, sizeof (*d_c),d_c ,N,h_c,N); // cp d_c ->c
printf ("Hasil Matrix Calculation :\n");
for(i=0;i<N;i ++){
for(j=0;j<N;j ++){
printf (" %7.0 f",c[ IDX2C (i,j,N )]); // print c after Sgemm
}
printf ("\n");
}
hipFree (d_a ); // free device memory
hipFree (d_b ); // free device memory
hipFree (d_c ); // free device memory
hipblasDestroy ( handle ); // destroy CUBLAS context
free (a); // free host memory
free (b); // free host memory
free (c); // free host memory
return 0 ;
}