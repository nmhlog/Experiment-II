#include "hip/hip_runtime.h"
/*
Matrix Multiplication in CUDA
Modified from :
https://github.com/lzhengchun/matrix-cuda/blob/master/matrix_cuda.cu
https://www.geeksforgeeks.org/strassens-matrix-multiplication/
Book : Programming Massively Parallel Processor Chapter 4
*/
#include <time.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#define TILE_WIDTH 25

__global__ void gpuMatrixMul(float *d_a , float *d_b, float *d_c,int N) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if ((Row < N) && (Col < N)) {
		float Pvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < N; ++k) {
			Pvalue += d_a[Row*N+k]*d_b[k*N+Col];
			}
		d_c[Row*N+Col] = Pvalue;
		}
}

void cpuMatrixMul(float *h_a, float *h_b, float *h_c, int N) {
    for (int i = 0; i < N; ++i) // Row
    {
        for (int j = 0; j < N; ++j) //Col
        {
            int tmp = 0.0;
            for (int h = 0; h < N; ++h)  // Row
            {
                tmp += h_a[i * N + h] * h_b[h * N + j];
            }
            h_c[i * N + j] = tmp;
        }
    }
}

__global__ void gpuTiledMatrixMul(float* d_M, float* d_N, float* d_P, int Width,int TILE_WIDTH) {
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	
	for (int ph = 0; ph < ceil(Width/(float)TILE_WIDTH); ++ph) {
	// Collaborative loading of d_M and d_N tiles into shared memory
	if ((Row< Width) && (ph*TILE_WIDTH+tx)< Width) Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
	if ((ph*TILE_WIDTH+ty)<Width && Col<Width) Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
	__syncthreads();
	
		for (int k = 0; k < TILE_WIDTH; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	if ((Row<Width) && (Col<Width)) d_P[Row*Width + Col] = Pvalue;
}

void verification(float *h_c,float *h_cc,int N,float denominator,float numerator){
	int all_ok = 1;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
           
            if(h_cc[i*N+ j] != h_c[i*N + j])
            {
                all_ok = 0;
            }
        }
    }

    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", numerator / denominator);
    }
    else
    {
        printf("incorrect results\n");
    }
}

void print_matrix(float *h_matrix,int N){
	for (int i = 0; i < N; ++i)
    {
		printf("[ ");
        for (int j = 0; j < N; ++j)
        {            
		printf("%f ",h_matrix[i*N+ j]);	
        }
		printf("] \n");
    }


}
int main(int argc, char const *argv[]){
	if (argc < 4) {
		printf("Required args: N(dimension),BlockSize(), k\n");
		exit(-1);
	}
    int N = atoi(argv[1]);
    int BLOCK_SIZE = atoi(argv[2]);  
	int const TILE_WIDTH = atoi(argv[3]);
    int nBytes = N*N*sizeof(float);
	float *h_a, *h_b, *h_c,*h_cc,*h_tcc;
	// Memory Allocation in Host
	h_a = (float *)malloc(nBytes);
	h_b = (float *)malloc(nBytes);
    h_c = (float *)malloc(nBytes);
    h_cc = (float *)malloc(nBytes);
    h_tcc = (float *)malloc(nBytes);
	//  Initialization of a and b Matrix
	for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
		  h_a[i * N + j] = 1.0;
		  h_b[i * N + j] = 2.0;
      }
	}
	// printf("Matrix A \n");
	// print_matrix(h_a,N);
	// printf("Matrix b \n");
	// print_matrix(h_a,N);
	// printf("\n");


	float gpu_elapsed_time_ms, cpu_elapsed_time_ms,gpu_tiled_elapsed_time_ms,gpu_cublas_elapsed_time_ms;
	// some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
	float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(float)*N*N);
    hipMalloc((void **) &d_b, sizeof(float)*N*N);
    hipMalloc((void **) &d_c, sizeof(float)*N*N);
    
	

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(float)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*N*N, hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    /*
    Cuda Matmul using Vanila Code
    */

	gpuMatrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c,N);
	hipMemcpy(h_c, d_c, sizeof(float)*N*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

	// compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", N,N,N,N, gpu_elapsed_time_ms);
    
    /*
    Matmul using Vanila Code in CPU
    */

    // start the CPU version
    hipEventRecord(start, 0);
    cpuMatrixMul(h_a, h_b, h_cc,N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", N,N,N,N, cpu_elapsed_time_ms);
	
    /*
    Cuda tiled Matmul 
    */
    hipEventRecord(start, 0);
	gpuTiledMatrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c,N,TILE_WIDTH);
	hipMemcpy(h_tcc, d_c, sizeof(float)*N*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_tiled_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", N,N,N,N, gpu_tiled_elapsed_time_ms);

    /* 
    Verification each output and calculated Speed up
    */
    printf("gpu to cpu speedup and verification \n");
    verification(h_c,h_cc,N,gpu_elapsed_time_ms,cpu_elapsed_time_ms);
    printf("tiled gpu to gpu speedup and verification \n");
    verification(h_tcc,h_c,N,gpu_tiled_elapsed_time_ms,gpu_elapsed_time_ms);
    printf("tiled gpu to cpu speedup and verification \n");
    verification(h_tcc,h_cc,N,gpu_tiled_elapsed_time_ms,cpu_elapsed_time_ms);

	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
